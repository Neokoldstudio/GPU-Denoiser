#include "hip/hip_runtime.h"
//------------------------------------------------------
// Prog    : Denoiser.cu
// auteur original  : Mignotte Max
// portage sur GPU : Godbert Paul
// date    :
// version : 1.0
// langage : CUDA C
// labo    : DIRO
// note    :
//------------------------------------------------------

//------------------------------------------------
// LIBRAIRIES ------------------------------------
//------------------------------------------------
#include <time.h>

//------------------------------------------------
// FICHIERS INCLUS -------------------------------
//------------------------------------------------
#include "Fonctions.h"

//------------------------------------------------
// DEFINITIONS -----------------------------------
//------------------------------------------------
#define NAME_VISUALISER "display "
#define NAME_IMG_IN "../Images/lena512"
#define NAME_IMG_OUT "../Images/lena512_Restored"
#define NAME_IMG_DEG "../Images/lena512_Degraded"
//------------------------------------------------
// PROTOTYPE DE FONCTIONS  -----------------------
//------------------------------------------------
//>Main Function
void DctDenoise(float **, float **, float **, int, int, float);

//>Gestion
void copy_matrix(float **, float **, int, int);
void FilteringDCT_8x8_(float **, float, int, int, float **, float ***);
void FilteringDCT_8x8(float **, float, int, int, float **, float ***);
void ZigZagThreshold(float, float *, int);
void copy_matrix_on_device(float *, float **, int, int);
void copy_matrix_on_host(float **, float *, int , int);
void copy_matrix_1d_to_2d(float*,float**,int,int);

__global__ void HardThreshold(float, float *, int);
__global__ void denoise_image(float *, float *, int, int, int, int);
__global__ void denoise_block(float *, float, int, int, int, float *, float *, void (*)(float, float*, int));

#define SIGMA_NOISE 30
#define NB_ITERATIONS 1
#define THRESHOLD 90
#define OVERLAP 2
#define HARD_THRESHOLD 1

#define ZOOM 1
#define QUIT 0

//------------------------------------------------
//------------------------------------------------
// FONCTIONS  ------------------------------------
//------------------------------------------------
//------------------------------------------------
//----------------------------------------------------------
// IterDctDenoise
//----------------------------------------------------------

__global__ void simple_kernel(float *input, float *output, int length, int width) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < length && idy < width) {
        int index = idy * width + idx;  // Corrected index calculation
        output[index] = input[index];   // Simply copy input to output
    }
}

void DctDenoise(float **DataDegraded, float *DataFiltered_d, float **Data, int lgth, int wdth, float Thresh)
{
    int k;
    int SizeWindow;
    char Name_img[NBCHAR];

    // Parameter
    SizeWindow = 8;

    // Info
    printf("\n  --------------------- ");
    printf("\n      IterDctDenoise ");
    printf("\n  ---------------------");
    printf("\n      Length:Width [%d][%d]", lgth, wdth);
    printf("\n      -----------------------");
    printf("\n      >> SigmaNoise = [%d]", SIGMA_NOISE);
    printf("\n      -----------------------");
    printf("\n      Threshold_Dct  > [%.1d]", THRESHOLD);
    printf("\n      Size Window    > [%d]", SizeWindow);
    printf("\n      Overlap        > [%d]", OVERLAP);
    printf("\n\n");

    // Allocation Memoire
    float*** mat3d = fmatrix_allocate_3d(SizeWindow * SizeWindow, lgth, wdth);
    float*   DataFilteredDst_d = fmatrix_allocate_2d_device(lgth, wdth);
    float**  DataFiltered_h = fmatrix_allocate_2d(lgth, wdth);

    // Init
    copy_matrix_on_device(DataFiltered_d, DataDegraded, lgth, wdth);

    // Define block size
    int blockSize = SizeWindow; 

    // Calculate grid dimensions
    int blocksX = (lgth + blockSize - 1) / blockSize;
    int blocksY = (wdth + blockSize - 1) / blockSize;

    // Set up the thread block and grid dimensions
    dim3 threadsPerBlock(blockSize, blockSize);
    dim3 blocksPerGrid(blocksX, blocksY);

    printf("      --------Kernel Called on :------\n");
    printf("          Threads Per Block: %d x %d\n", threadsPerBlock.x, threadsPerBlock.y);
    printf("          Blocks Per Grid: %d x %d\n", blocksPerGrid.x, blocksPerGrid.y);
    printf("      --------------------------------\n");

    // Debug print before denoising
    // Launch kernel for denoising
    for (k = 0; k < NB_ITERATIONS; k++)
    {
        for(int torioidalShiftY = 0; torioidalShiftY < 8; torioidalShiftY++)
        {
            for(int torioidalShiftX = 0; torioidalShiftX < 8; torioidalShiftX++)
            {
                // Toroidal Shift
                float *DataShifted_d;
                hipMalloc((void**)&DataShifted_d, lgth * wdth * sizeof(float));
                
                int shiftX = torioidalShiftX * OVERLAP;
                int shiftY = torioidalShiftY * OVERLAP;

                ToroidalShift<<<blocksPerGrid, threadsPerBlock>>>(DataShifted_d, DataFiltered_d, lgth, wdth, shiftX, shiftY);
                hipDeviceSynchronize();
                //Launch Discrete Cosine Transform
                CUDA_DCT8x8<<<blocksPerGrid, threadsPerBlock>>>(DataFilteredDst_d, wdth, DataShifted_d);
                hipDeviceSynchronize();

                // Launch Quantization kernel (here, a simple Hardthreshold)
                //HardThreshold<<<blocksPerGrid, threadsPerBlock>>>(SIGMA_NOISE, DataFilteredDst_d, lgth);
                //hipDeviceSynchronize();

                // Launch Inverse Discrete Cosine Transform
                //CUDA_IDCT8x8<<<blocksPerGrid, threadsPerBlock>>>(DataFilteredDst_d, wdth, DataFilteredDst_d);
                //hipDeviceSynchronize();

                // Allocate host buffer for the current toroidal shift
                float* tempBuffer = (float*)malloc(lgth * wdth * sizeof(float));

                // Copy data from device to the host buffer
                hipMemcpy(tempBuffer, DataFilteredDst_d, lgth * wdth * sizeof(float), hipMemcpyDeviceToHost);

                // Calculate the offset for the current toroidal shift
                int offset = (torioidalShiftY * 8 + torioidalShiftX);

                // Copy data from the host buffer to the appropriate location in the 3D host array
                for (int i = 0; i < lgth; i++)
                {
                    for (int j = 0; j < wdth; j++)
                    {
                        mat3d[offset][i][j] = tempBuffer[i * wdth + j];
                    }
                }

                // Free the temporary host buffer
                free(tempBuffer);
            }
        }
    }

    hipMemcpy(DataFiltered_d, DataFilteredDst_d, lgth*wdth*sizeof(float), hipMemcpyDeviceToDevice);

    for (int i = 0; i < lgth; i++)
    {
        for (int j = 0; j < wdth; j++)
        {
            float temp = 0.0;
            double nb = 0.0;
            for (k = 0; k < 64; k++)
            {
                if (mat3d[k][i][j] > 0.0)
                {
                    nb++;
                    temp += mat3d[k][i][j];
                }
            }
            if (nb)
            {
                temp /= nb;
                DataFiltered_h[i][j] = temp;
            }
        }
    }

    copy_matrix_on_device(DataFiltered_d, DataFiltered_h, lgth, wdth);

    if(DataFiltered_h)
        free_fmatrix_2d(DataFiltered_h);
    free_matrix_device(DataFilteredDst_d);
}
//---------------//
//--- GESTION ---//
//---------------//
//----------------------------------------------------------
// copy matrix
//----------------------------------------------------------
void copy_matrix(float **mat1, float **mat2, int lgth, int wdth)
{
    int i, j;

    for (i = 0; i < lgth; i++)
        for (j = 0; j < wdth; j++)
            mat1[i][j] = mat2[i][j];
}

void copy_matrix_1d_to_2d(float *mat1, float **mat2, int lgth, int wdth)
{
    int i, j;

    for (i = 0; i < lgth; i++)
        for (j = 0; j < wdth; j++)
            mat2[i][j] = mat1[i * wdth + j];
}

void copy_matrix_2d_to_1d(float **mat1, float *mat2, int lgth, int wdth)
{
    int i, j;

    for (i = 0; i < lgth; i++)
        for (j = 0; j < wdth; j++)
            mat2[i * wdth + j] = mat1[i][j];
}

void copy_matrix_on_device(float *mat1, float **mat2, int lgth, int wdth)
{
    float* buff = new float[lgth * wdth];

    for (int i = 0; i < lgth; i++)
        for (int j = 0; j < wdth; j++)
            buff[i * wdth + j] = mat2[i][j];

    size_t size = lgth * wdth * sizeof(float);

    hipMemcpy(mat1, buff, size, hipMemcpyHostToDevice);

    hipFree(buff);
}

void copy_matrix_on_host(float **mat1, float *mat2, int lgth, int wdth)
{
    float *buff = (float *)malloc(lgth * wdth * sizeof(float));

    hipMemcpy(buff, mat2, lgth * wdth * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < lgth; i++)
        for (int j = 0; j < wdth; j++)
            mat1[i][j] = buff[wdth * i + j];

    free(buff);
}

//----------------------------------------------------------
//  DCT thresholding
//----------------------------------------------------------
__global__ void HardThreshold(float sigma, float *coef, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < N)
    {
        int index = i + N * j;

        if (fabs(coef[index]) < sigma)
        {
            coef[index] = 0.0;
        }
    }
}
//----------------------------------------------------------
//  DCT ZigZag thresholding
//----------------------------------------------------------
__device__ void ZigZagThreshold(float sigma, float *coef, int N)
{
    int result[8][8];
    int i = 0;
    int j = 0;
    int d = -1;
    int start = 0;
    int end = (N * N) - 1;

    //>ZigZag Matrix
    do
    {
        result[i][j] = start++;
        result[N - i - 1][N - j - 1] = end--;

        i += d;
        j -= d;
        if (i < 0)
        {
            i++;
            d = -d;
        }
        else if (j < 0)
        {
            j++;
            d = -d;
        }
    } while (start < end);
    if (start == end)
        result[i][j] = start;

    //>Seuillage
    for (i = 0; i < N; i++)
        for (j = 0; j < N; j++)
            if (result[i][j] >= sigma)
                coef[i + N * j] = 0.0;
}


void usage(const char* programName)
{
    printf("Usage: %s [<input_image.pgm>]\n", programName);
    printf("If no input image is provided, the default image will be used.\n");
}
//---------------------------------------------------------
//---------------------------------------------------------
// PROGRAMME PRINCIPAL   ----------------------------------
//---------------------------------------------------------
//---------------------------------------------------------

int main(int argc, char** argv)
{
    char* inputImage;

    if (argc == 1)
    {
        inputImage = NAME_IMG_IN;
    }
    else if (argc == 2)
    {
        inputImage = argv[1];
    }
    else
    {
        usage(argv[0]);
        return 1;
    }

    printf("-------------Current GPU--------------\n");

    hipSetDevice(0);
    int deviceId;
    hipGetDevice(&deviceId);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);
    printf("Device Name: %s\n", deviceProp.name);
    printf("Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
    printf("Max Threads Per Block: %d\n", deviceProp.maxThreadsPerBlock);
    printf("Max Grid Size: %d x %d x %d\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    printf("Max Threads Dimension: %d x %d x %d\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("--------------------------------------\n");

    int length, width;
    char BufSystVisuImg[NBCHAR];

    //>Lecture Image
    float** Img = LoadImagePgm(inputImage, &length, &width);

    if (!Img)
    {
        printf("Error loading image %s\n", inputImage);
        return 1;
    }

    //>CPU Memory Allocation
    float** ImgDegraded = fmatrix_allocate_2d(length, width);
    float** ImgDenoised = fmatrix_allocate_2d(length, width);

    //>GPU Memory Allocation
    float* ImgDenoised_d = fmatrix_allocate_2d_device(length, width);

    copy_matrix(ImgDegraded, Img, length, width);
    add_gaussian_noise(ImgDegraded, length, width, SIGMA_NOISE * SIGMA_NOISE);

    printf("\n  Info Noise");
    printf("\n  ---------------------");
    printf("\n  Before Denoising :");
    printf("\n      > MSE = [%.2f]", computeMMSE(ImgDegraded, Img, length));

    clock_t start = clock();
    DctDenoise(ImgDegraded, ImgDenoised_d, Img, length, width, THRESHOLD);
    clock_t end = clock();

    double duration = (double)(end - start) / CLOCKS_PER_SEC * 1000.0;

    copy_matrix_on_host(ImgDenoised, ImgDenoised_d, length, width);

    printf("\n  ---------------------");
    printf("\n  After Denoising :");
    printf("\n      > MSE = [%.2f]", computeMMSE(ImgDenoised, Img, length));
    printf("\n  ---------------------");
    printf("\n  Duration :");
    printf("\n  Temps d'exécution de DctDenoise : %.2f ms\n", duration);
    printf("\n  ---------------------");

    SaveImagePgm(NAME_IMG_DEG, ImgDegraded, length, width);
    SaveImagePgm(NAME_IMG_OUT, ImgDenoised, length, width);

    strcpy(BufSystVisuImg, NAME_VISUALISER);
    strcat(BufSystVisuImg, inputImage);
    strcat(BufSystVisuImg, ".pgm&");
    printf("\n > %s", BufSystVisuImg);
    system(BufSystVisuImg);

    strcpy(BufSystVisuImg, NAME_VISUALISER);
    strcat(BufSystVisuImg, NAME_IMG_DEG);
    strcat(BufSystVisuImg, ".pgm&");
    printf("\n > %s", BufSystVisuImg);
    system(BufSystVisuImg);

    strcpy(BufSystVisuImg, NAME_VISUALISER);
    strcat(BufSystVisuImg, NAME_IMG_OUT);
    strcat(BufSystVisuImg, ".pgm&");
    printf("\n > %s", BufSystVisuImg);
    system(BufSystVisuImg);

    if (Img) free_fmatrix_2d(Img);
    if (ImgDegraded) free_fmatrix_2d(ImgDegraded);
    if (ImgDenoised) free_fmatrix_2d(ImgDenoised);
    free_matrix_device(ImgDenoised_d);

    printf("\n C'est fini... \n");
    return 0;
}